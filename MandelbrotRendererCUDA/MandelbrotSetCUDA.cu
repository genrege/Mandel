#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "MandelbrotSetCUDA.h"

__global__ void kernel_mbrot(double x0, double x1, double y0, double y1, int wx, int wy, double w, double h, int max_iter, unsigned int* r, unsigned* p, unsigned palette_index)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= wx)
        return;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (iy >= wy)
        return;

    const double cr = x0 + w * ix;
    const double ci = y1 - h * iy;

    double zr = 0.0;
    double zi = 0.0;

    double zr2 = zr * zr;
    double zi2 = zi * zi;
    
    int iter = 0;
    while (iter < max_iter && (zr2 + zi2) <= 4.0)
    {
        zi = (zr + zr) * zi + ci;
        zr = zr2 - zi2 + cr;

        zr2 = zr * zr;
        zi2 = zi * zi;

        ++iter;
    }
    const auto idx = ix + wx * iy;
    if (p == nullptr)
        r[idx] = iter;
    else
        r[idx] = iter >= max_iter ? 0 : p[(iter + palette_index) % max_iter];
}

__global__ void kernel_julia(double x0, double x1, double y0, double y1, double kr, double ki, double w, double h, int wx, int wy, int max_iter, unsigned* r, unsigned* p, unsigned palette_index)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= wx)
        return;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (iy >= wy)
        return;

    const double cr = x0 + w * ix;
    const double ci = y1 - h * iy;

    double zr = cr;
    double zi = ci;

    double zr2 = zr * zr;
    double zi2 = zi * zi;

    int iter = 0;
    while (iter < max_iter && (zr2 + zi2) < 4.0)
    {
        zi = (zr + zr) * zi + ki;
        zr = zr2 - zi2 + kr;

        zr2 = zr * zr;
        zi2 = zi * zi;

        ++iter;
    }
    const int idx = ix + wx * iy;
    if (p == nullptr)
        r[idx] = iter;
    else
        r[idx] = iter >= max_iter ? 0 : p[(iter + palette_index) % max_iter];
}


mbrot_cuda::mbrot_cuda() : m_dev_r(nullptr), m_csize(0)
{
    hipSetDevice(0);
}

mbrot_cuda::~mbrot_cuda()
{
    hipFree(m_dev_r);
}

unsigned int* mbrot_cuda::alloc_cuda(int size)
{
    if (size != m_csize)
    {
        hipFree(m_dev_r);
        hipMalloc(&m_dev_r, size);
        m_csize = size;
    }
    return m_dev_r;
}

unsigned int* mbrot_cuda::alloc_palette(int size)
{
    if (size != m_psize)
    {
        hipFree(m_dev_p);
        hipMalloc(&m_dev_p, size);
        m_psize = size;
    }
    return m_dev_p;
}

void mbrot_cuda::render_mbrot(int wx, int wy, double x0, double x1, double y0, double y1, int max_iter, unsigned int* r, unsigned* palette, unsigned palette_index)
{
    const double w = (x1 - x0) / double(wx);
    const double h = (y1 - y0) / double(wy);

    const int gs = 32;
    int extra = (wx % gs == 0) ? 0 : 1;
    dim3 threads(gs, gs);
    dim3 blocks(wx / gs + extra, wy / gs + extra);

    auto* dev_r = alloc_cuda(sizeof(unsigned int) * wx * wy);
    unsigned* dev_p = nullptr;
    if (palette != nullptr)
    {
        dev_p = alloc_palette(sizeof(unsigned) * (1 + max_iter));
        hipMemcpy(dev_p, palette, sizeof(unsigned) * (1 + max_iter), hipMemcpyKind::hipMemcpyHostToDevice);
    }

    kernel_mbrot << <blocks, threads>> > (x0, x1, y0, y1, wx, wy, w, h, max_iter, dev_r, dev_p, palette_index);
    hipMemcpy(r, dev_r, sizeof(unsigned int) * wx * wy, hipMemcpyKind::hipMemcpyDeviceToHost);
}

void mbrot_cuda::render_julia(int wx, int wy, double x0, double x1, double y0, double y1, double kr, double ki, int max_iter, unsigned int* r, unsigned* palette, unsigned palette_index)
{
    const double w = (x1 - x0) / double(wx);
    const double h = (y1 - y0) / double(wy);

    const int gs = 32;
    int extra = (wx % gs == 0) ? 0 : 1;
    dim3 threads(gs, gs);
    dim3 blocks(wx / gs + extra, wy / gs + extra);

    auto* dev_r = alloc_cuda(sizeof(unsigned int) * wx * wy);
    unsigned* dev_p = nullptr;
    if (palette != nullptr)
    {
        dev_p = alloc_palette(sizeof(unsigned) * (1 + max_iter));
        hipMemcpy(dev_p, palette, sizeof(unsigned) * (1 + max_iter), hipMemcpyKind::hipMemcpyHostToDevice);
    }

    kernel_julia << <blocks, threads>> > (x0, x1, y0, y1, kr, ki, w, h, wx, wy, max_iter, dev_r, dev_p, palette_index);
    hipMemcpy(r, dev_r, sizeof(unsigned int) * wx * wy, hipMemcpyKind::hipMemcpyDeviceToHost);
}

