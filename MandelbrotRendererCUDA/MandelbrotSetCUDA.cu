#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "MandelbrotSetCUDA.h"

__global__ void kernel_mbrot(double x0, double x1, double y0, double y1, int wx, int wy, double w, double h, int max_iter, unsigned int* r)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix > wx)
        return;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (iy > wy)
        return;

    const double c_re = x0 + w * ix;
    const double c_im = y1 - h * iy;

    int iter = 0;

    double  z_re = 0.0;
    double z_im = 0.0;

    double zr2 = z_re * z_re;
    double zi2 = z_im * z_im;
    while (iter < max_iter && (zr2 + zi2) < 4.0)
    {
        const double z2_re = zr2 - zi2 + c_re;
        z_im = 2.0 * z_re * z_im + c_im;
        z_re = z2_re;
        ++iter;

        zr2 = z_re * z_re;
        zi2 = z_im * z_im;
    }
    const auto idx = ix + wx * iy;
    r[idx] = iter;
}

__global__ void kernel_julia(double x0, double x1, double y0, double y1, double kr, double ki, double w, double h, int wx, int wy, int max_iter, unsigned int* r)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix > wx)
        return;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (iy > wy)
        return;

    const double c_re = x0 + w * ix;
    const double c_im = y1 - h * iy;

    double z_re = c_re;
    double z_im = c_im;

    double zr2 = z_re * z_re;
    double zi2 = z_im * z_im;

    int iter = 0;
    while (iter < max_iter && (zr2 + zi2) < 4.0)
    {
        double z2_re = zr2 - zi2 + kr;
        z_im = 2.0 * z_re * z_im + ki;
        z_re = z2_re;
        ++iter;

        zr2 = z_re * z_re;
        zi2 = z_im * z_im;
    }
    const int idx = ix + wx * iy;
    r[idx] = iter;
}


mbrot_cuda::mbrot_cuda() : m_dev_r(nullptr), m_csize(-1)
{
    hipSetDevice(0);
}

unsigned int* mbrot_cuda::alloc_cuda(int size)
{
    if (size != m_csize)
    {
        hipFree(m_dev_r);
        hipMallocManaged((void**)&m_dev_r, size);
        m_csize = size;
    }
    return m_dev_r;
}

void mbrot_cuda::render_mbrot(double x0, double x1, double y0, double y1, int wx, int wy, int max_iter, unsigned int* r)
{
    const double w = (x1 - x0) / double(wx);
    const double h = (y1 - y0) / double(wy);

    const int gs = 96;
    dim3 grid(gs, gs);
    dim3 block(wx / gs + 1, wy / gs + 1);
    kernel_mbrot << <grid, block >> > (x0, x1, y0, y1, wx, wy, w, h, max_iter, r);
    hipDeviceSynchronize();
}

void mbrot_cuda::render_julia(double x0, double x1, double y0, double y1, double kr, double ki, int wx, int wy, int max_iter, unsigned int* r)
{
    const double w = (x1 - x0) / double(wx);
    const double h = (y1 - y0) / double(wy);

    const int gs = 96;
    dim3 grid(gs, gs);
    dim3 block(wx / gs + 1, wy / gs + 1);
    kernel_julia << <grid, block >> > (x0, x1, y0, y1, kr, ki, w, h, wx, wy, max_iter, r);
    hipDeviceSynchronize();
}
