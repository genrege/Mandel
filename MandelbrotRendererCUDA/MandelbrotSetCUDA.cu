#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "MandelbrotSetCUDA.h"
#include <stdio.h>
#include <Windows.h>

__global__ void kernel_mbrot(double x0, double x1, double y0, double y1, int wx, int wy, double w, double h, int max_iter, unsigned int* r)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= wx)
        return;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (iy >= wy)
        return;

    const double cr = x0 + w * ix;
    const double ci = y1 - h * iy;

    int iter = 0;

    double zr = 0.0;
    double zi = 0.0;

    double zr2 = zr * zr;
    double zi2 = zi * zi;
    
    while (iter < max_iter && (zr2 + zi2) < 4.0)
    {
        zi = (zr + zr) * zi + ci;
        zr = zr2 - zi2 + cr;

        zr2 = zr * zr;
        zi2 = zi * zi;

        ++iter;
    }
    const auto idx = ix + wx * iy;
    r[idx] = iter;
}

__global__ void kernel_julia(double x0, double x1, double y0, double y1, double kr, double ki, double w, double h, int wx, int wy, int max_iter, unsigned int* r)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= wx)
        return;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (iy >= wy)
        return;

    const double cr = x0 + w * ix;
    const double ci = y1 - h * iy;

    double zr = cr;
    double zi = ci;

    double zr2 = zr * zr;
    double zi2 = zi * zi;

    int iter = 0;
    while (iter < max_iter && (zr2 + zi2) < 4.0)
    {
        zi = (zr + zr) * zi + ki;
        zr = zr2 - zi2 + kr;

        zr2 = zr * zr;
        zi2 = zi * zi;

        ++iter;
    }
    const int idx = ix + wx * iy;
    r[idx] = iter;
}


mbrot_cuda::mbrot_cuda() : m_dev_r(nullptr), m_csize(0)
{
    hipSetDevice(0);
}

mbrot_cuda::~mbrot_cuda()
{
    hipFree(m_dev_r);
}

unsigned int* mbrot_cuda::alloc_cuda(int size)
{
    if (size != m_csize)
    {
        hipFree(m_dev_r);
        hipMalloc(&m_dev_r, size);
        m_csize = size;
    }
    return m_dev_r;
}

void mbrot_cuda::render_mbrot(double x0, double x1, double y0, double y1, int wx, int wy, int max_iter, unsigned int* r)
{
    const double w = (x1 - x0) / double(wx);
    const double h = (y1 - y0) / double(wy);

    const int gs = 72;
    dim3 blocks(gs, gs);
    dim3 threads(wx / gs + 1, wy / gs + 1);

    auto * dev_r = alloc_cuda(sizeof(unsigned int) * wx * wy);
    kernel_mbrot << <blocks, threads>> > (x0, x1, y0, y1, wx, wy, w, h, max_iter, dev_r);
    hipMemcpy(r, dev_r, sizeof(unsigned int) * wx * wy, hipMemcpyKind::hipMemcpyDeviceToHost);
}

void mbrot_cuda::render_julia(double x0, double x1, double y0, double y1, double kr, double ki, int wx, int wy, int max_iter, unsigned int* r)
{
    const double w = (x1 - x0) / double(wx);
    const double h = (y1 - y0) / double(wy);

    const int gs = 96;
    dim3 grid(gs, gs);
    dim3 block(wx / gs + 1, wy / gs + 1);

    auto* dev_r = alloc_cuda(sizeof(unsigned int) * wx * wy);
    kernel_julia << <grid, block >> > (x0, x1, y0, y1, kr, ki, w, h, wx, wy, max_iter, dev_r);
    hipMemcpy(r, dev_r, sizeof(unsigned int) * wx * wy, hipMemcpyKind::hipMemcpyDeviceToHost);
}
